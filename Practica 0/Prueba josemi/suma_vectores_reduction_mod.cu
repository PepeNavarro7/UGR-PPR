
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "iostream"
#include <time.h>
#include <chrono>

int N;

#define Bsize_addition 256
#define Bsize_minimum   128


using namespace std;
using namespace std::chrono;

//**************************************************************************
// Vector addition kernel
//**************************************************************************
__global__ void add_arrays_gpu( float *in1, float *in2, float *out, int N)
{
int idx=blockIdx.x*blockDim.x+threadIdx.x;
if (idx<N) 
     out[idx]=in1[idx]+in2[idx];
}
//**************************************************************************



//**************************************************************************
// Vector minimum  kernel
//**************************************************************************
__global__ void reduceMin(float * V_in, float * V_out, const int N) {
	extern __shared__ float sdata[];

	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = ((i < N) ? V_in[i] : 100000000.0f);
	sdata[tid + blockDim.x] = (((i + N/2) < N) ? V_in[i + N/2] : 100000000.0f);
	__syncthreads();

	for(int s = blockDim.x/2; s > 0; s >>= 1){
	  if (tid < s)
	        sdata[tid]=min(sdata[tid],sdata[tid+s]); 	
	  __syncthreads();
	}
	if (tid == 0) 
           V_out[blockIdx.x] = sdata[0];
}




//**************************************************************************
int main(int argc, char **argv)
//**************************************************************************

{
if (argc != 2)
{
	cout << "Error: numero de parametros incorrecto" << endl;
	cout << "./ejecutable <N>" << endl;
	exit(-1);
}

N = atoi(argv[1]);
srand(time(NULL));
/* pointers to host memory */
float *a, *b, *c;
/* pointers to device memory */
float *a_d, *b_d, *c_d;
int i;

/* Allocate arrays a, b and c on host*/
a = (float*) malloc(N*sizeof(float));
b = (float*) malloc(N*sizeof(float));
c = (float*) malloc(N*sizeof(float));

/* Allocate arrays a_d, b_d and c_d on device*/
hipMalloc ((void **) &a_d, sizeof(float)*N);
hipMalloc ((void **) &b_d, sizeof(float)*N);
hipMalloc ((void **) &c_d, sizeof(float)*N);

/* Initialize arrays a and b */
for (i=0; i<N;i++)
{
a[i]= (float) (rand()%1000); 
b[i]= (float) (rand()%45);
}


/* Copy data from host memory to device memory */
hipMemcpy(a_d, a, sizeof(float)*N, hipMemcpyHostToDevice);
hipMemcpy(b_d, b, sizeof(float)*N, hipMemcpyHostToDevice);

/* Compute the execution configuration */
dim3 dimBlock(Bsize_addition);
dim3 dimGrid ( ceil((float(N)/(float)dimBlock.x)) );

/* Add arrays a and b, store result in c */
add_arrays_gpu<<< dimGrid, dimBlock >>>(a_d, b_d, c_d, N);

//**************************************************
// Block reduction on GPU to obtain partial minimums
//**************************************************
dim3 threadsPerBlock(Bsize_minimum / 2);
dim3 numBlocks( ceil ((((float) N) / 2.0f)/threadsPerBlock.x));

// Minimum vector on CPU
float * vmin;
vmin = (float*) malloc(numBlocks.x*sizeof(float));

// Minimum vector  to be computed on GPU
float *vmin_d; 
hipMalloc ((void **) &vmin_d, sizeof(float)*numBlocks.x);

int smemSize = threadsPerBlock.x*sizeof(float)*2;

auto start_time = high_resolution_clock::now();
// Kernel launch to compute Minimum Vector
reduceMin<<<numBlocks, threadsPerBlock, smemSize>>>(c_d,vmin_d, N);
hipDeviceSynchronize();

auto end_time = high_resolution_clock::now();
auto duration_ns = duration_cast<nanoseconds>(end_time - start_time).count();

cout << N << " " << duration_ns << endl;

/* Copy data from device memory to host memory */
hipMemcpy(vmin, vmin_d, numBlocks.x*sizeof(float),hipMemcpyDeviceToHost);
hipMemcpy(c, c_d, sizeof(float)*N,hipMemcpyDeviceToHost);

// Perform final reduction in CPU
float min_gpu = 10000000.0f;
// cout<<"Performing final reduction of the partial results on CPU:"<<endl;
for (int i=0; i<numBlocks.x; i++) 
{
  min_gpu =min(min_gpu,vmin[i]); 
  // cout<<"vmin["<<i<<"]="<<vmin[i]<<"    ";
}

// cout<<endl<<"... Minimum on GPU ="<<min_gpu<<"               ";


//***********************
// Compute minimum on CPU
//***********************
float min_cpu=1000000.0f;
for (i=0; i<N;i++)
{
 //ut<<"c["<<i<<"]="<<c[i]<<endl;
 min_cpu=min(min_cpu, c[i]);
}

// cout<<".... Minimum on CPU="<< min_cpu<<endl;


/* Free the memory */
free(a); free(b); free(c);free(vmin);
hipFree(a_d); hipFree(b_d);hipFree(c_d);hipFree(vmin_d);
}
