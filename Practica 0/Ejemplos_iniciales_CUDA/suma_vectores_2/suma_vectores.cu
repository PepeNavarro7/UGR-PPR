
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>


using namespace std;

const int N=150;

__global__ void add_arrays_gpu( float *in1, float *in2, float *out, int Ntot){
	int idx=blockIdx.x*blockDim.x+threadIdx.x; // indice de la hebra
	int i=idx*2; // Cada hebra idx se ocupa del lugar i y de i+1
	if (i<Ntot){
		out[i]=in1[i]+in2[i];
		printf("Block %i,   Thread %i::  C[%i] = A[%i]+B[%i] = %f \n",blockIdx.x,threadIdx.x,i,i,i,out[i] ); 
		i++;
		if(i<Ntot){
			out[i]=in1[i]+in2[i];
			printf("Block %i,   Thread %i::  C[%i] = A[%i]+B[%i] = %f \n",blockIdx.x,threadIdx.x,i,i,i,out[i] );
		}
	}
}

int main(int argc, char* argv[])
{
  /* pointers to host memory */
  float *a, *b, *c;
  /* pointers to device memory */
  float *a_d, *b_d, *c_d;

  /* Allocate arrays a, b and c on host*/
  a = (float*) malloc(N*sizeof(float));
  b = (float*) malloc(N*sizeof(float));
  c = (float*) malloc(N*sizeof(float));

  /* Allocate arrays a_d, b_d and c_d on device*/
  hipMalloc ((void **) &a_d, sizeof(float)*N);
  hipMalloc ((void **) &b_d, sizeof(float)*N);
  hipMalloc ((void **) &c_d, sizeof(float)*N);

  /* Initialize arrays a and b */
  for (int i=0; i<N;i++){
    a[i]= (float) i;
    b[i]= -(float) i;
  }

  /* Copy data from host memory to device memory */
  hipMemcpy(a_d, a, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, sizeof(float)*N, hipMemcpyHostToDevice);

  /* Compute the execution configuration */
  int block_size=64; // siempre multiplo de 32
  dim3 dimBlock(block_size);
  dim3 dimGrid ( ceil((float(N)/(float)dimBlock.x/2.0)) ); // Entre 2 porque cada hebra hace 2 operaciones

  /* Add arrays a and b, store result in c */
  add_arrays_gpu<<< dimGrid, dimBlock >>>(a_d, b_d, c_d, N);

  /* Copy data from deveice memory to host memory */
  hipMemcpy(c, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

  /* Print c */
  cout<<endl<<endl<<"C = { ";
  for (int i=0; i<N;i++)
     cout<<"  "<<c[i];
  cout<<" }"<<endl;   

  // Free the memory
  free(a); free(b); free(c);
  hipFree(a_d); hipFree(b_d);hipFree(c_d);

}
